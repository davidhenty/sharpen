
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__device__
double filter(int d, int i, int j);

__global__
void dosharpenpixel(int nx, int ny, int d, 
                    double *convolution, double *fuzzyPadded)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int idx, idxp, idxppk;

  if (i < nx && j < ny)
    {
      idx  = i*ny + j;
      idxp = (i+d)*(ny+2*d) + (j+d);

      for (int k= -d; k <= d; k++)
        {
          idxppk = idxp + k*(ny+2*d);

          for (int l= -d; l <= d; l++)
            {
                  convolution[idx] =   convolution[idx]
                                     + filter(d,k,l)*fuzzyPadded[idxppk+l];
            }
        }
    }
}
