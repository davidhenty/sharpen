
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__device__
double filter(int d, int i, int j);

__global__
void dosharpenpixel(int nx, int ny, int d, 
                    double *convolution, double *fuzzyPadded)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int idx, idxp, idxppk;

  if (i < nx && j < ny)
    {
      idx  = i*ny + j;
      idxp = (i+d)*(ny+2*d) + (j+d);

      for (int k= -d; k <= d; k++)
        {
          idxppk = idxp + k*(ny+2*d);

          for (int l= -d; l <= d; l++)
            {
                  convolution[idx] =   convolution[idx]
                                     + filter(d,k,l)*fuzzyPadded[idxppk+l];
            }
        }
    }
}

__global__
void dosharpenpixelinline(int nx, int ny, int d, 
                          double *convolution, double *fuzzyPadded)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int idx, idxp, idxppk;
  double fval;

  double rd4sq, rsq, sigmad4sq, sigmasq, x, y, delta;

  int d4 = 4;

  double sigmad4 = 1.4;
  double filter0 = -40.0;

  rd4sq = d4*d4;
  rsq   = d*d;

  sigmad4sq = sigmad4*sigmad4;
  sigmasq   = sigmad4sq * (rsq/rd4sq);

  if (i < nx && j < ny)
    {
      idx  = i*ny + j;
      idxp = (i+d)*(ny+2*d) + (j+d);

      /*      if ((i < nx/2) && (j < ny/2) || (i >= nx/2) && (j >= ny/2))
        {
          convolution[idx] = 127;
        }
      else
        {
          convolution[idx] = 255;
          } */
      
      for (int k= -d; k <= d; k++)
        {
          idxppk = idxp + k*(ny+2*d);

          for (int l= -d; l <= d; l++)
            {
              x = (double) k;
              y = (double) l;

              rsq = x*x + y*y;

              delta = rsq/(2.0*sigmasq);

              fval = filter0 * (1.0-delta) * exp(-delta);

              //              printf("i, j, k, l, idx, idxppk+l = %d, %d, %d, %d, %d, %d, convolution[idx], filter(d,k,l), fuzzyPadded[idxppk+l] = %f, %f, %f\n", i, j, k, l, idx, idxppk+l, convolution[idx], fval, fuzzyPadded[idxppk+l]);

                  convolution[idx] = convolution[idx]
                               + fval*fuzzyPadded[idxppk+l];
            }
        }
    }
}
