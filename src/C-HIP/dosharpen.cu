#include "hip/hip_runtime.h"
/*  Function to sharpen an image by convolving with a filter function. The
 *  filter is a combination of a Gaussian (to remove noise) and a Laplacian
 *  (to detect the edges). Input and output is via Portable Grey Map (PGM)
 *  files - note that the input file must have a specific header format.
 *
 *  In this version of the program the image processing is
 *  parallelised using CUDA for a GPU device. The CPU reads in the
 *  fuzzy image and stores it in host memory which is then copied to
 *  device memory on the GPU. CUDA kernels are launched on the GPU and
 *  the convolution computation is distributed over all CUDA threads.
 *  Finally, the CPU copies the result back to host memory, adds the
 *  convolution result to the fuzzy image and writes the resulting
 *  sharp image to file.
 *  
 *  David Henty, EPCC, September 2009
 *  Arno Proeme, EPCC, March 2013 (minor modifications)
 *  Dominic Sloan-Murphy, EPCC, June 2014 (improved consistency with other versions)
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utilities.h"
#include "sharpen.h"

__global__ void dosharpenpixel(int nx, int ny, int d,
                    double *convolution, double *fuzzyPadded);

void dosharpen(char *infile, int nx, int ny, int verbose)
{
  int d = 8;
  /* Sets the linear range of the sharpen filter as measured from any given pixel:
     only pixels within a (2d+1)*(2d+1) square centered on the pixel are used to 
     compute its new value. */

  double  norm = (2*d-1)*(2*d-1);
  double scale = 2.0;
  
  int xpix, ypix;
  
  int i, j;
  double tstart, tstop, time;
  
  int fuzzy[nx][ny];                   /* Will store the fuzzy input image when it is first read in from file                        */
  double fuzzyPadded[nx+2*d][ny+2*d];  /* Will store the fuzzy input image plus additional border padding                            */
  double convolution[nx][ny];          /* Will store the convolution of the filter with the fuzzy image                              */
  double sharp[nx][ny];                /* Will store the sharpened image obtained by adding the  convolution to the fuzzy image      */
  double sharpCropped[nx-2*d][ny-2*d]; /* Will store the sharpened image cropped to remove a border layer distorted by the algorithm */
  
  double *d_fuzzyp, *d_conv;

  char outfile[] = "sharpened.pgm";
  
  /* Initialise image arrays */
  for (i=0; i < nx; i++)
    {
      for (j=0; j < ny; j++)
        {
          fuzzy[i][j] = 0;
          sharp[i][j] = 0.0;
          convolution[i][j] = 0.0;
        }
    }

  if (verbose)
    {
      printf("Using a filter of size %d x %d\n", 2*d+1, 2*d+1);
      printf("\n");

      printf("Reading image file: %s\n", infile);
      fflush(stdout);
    }
       
    pgmread(infile, fuzzy, nx, ny, &xpix, &ypix);

  if (verbose)
    {
      printf("... done\n\n");
      fflush(stdout);
    }  

  if (xpix == 0 || ypix == 0 || nx != xpix || ny != ypix)
    {
      printf("Error reading %s\n", infile);
      fflush(stdout);
      exit(-1);
    }
  
  /* Initialise image array */
  for (i=0; i < nx+2*d; i++)
    {
      for (j=0; j < ny+2*d; j++)
        {
          fuzzyPadded[i][j] = 0.0;
        }
    }
  
  /* Transfer fuzzy image into padded array */
  for (i=0; i < nx; i++)
    { 
      for (j=0; j < ny; j++)
        {
          fuzzyPadded[i+d][j+d] = fuzzy[i][j];
        }
    }

  // Allocate CUDA memory
  hipMalloc((void **) &d_conv,   nx*ny*sizeof(double));
  hipMalloc((void **) &d_fuzzyp, (nx+2*d)*(ny+2*d)*ny*sizeof(double));

  // Copy

  hipMemcpy(d_conv, convolution, nx*ny*sizeof(double),
             hipMemcpyHostToDevice);

  hipMemcpy(d_fuzzyp, fuzzyPadded, (nx+2*d)*(ny+2*d)*sizeof(double),
             hipMemcpyHostToDevice);


  dim3 nthread = {16, 16, 1}; // 256 in a 16x16 grid
  dim3 nblock  = {(nx+nthread.x-1)/nthread.x, (ny+nthread.y-1)/nthread.y, 1};

  if (verbose)
    {
      printf("thread grid = %d x %d\n", nthread.x, nthread.y);
      printf("block  grid = %d x %d\n", nblock.x, nblock.y);

      printf("\nStarting calculation ...\n");
    }

  tstart = wtime();

  /* Start of parallel region where filter is applied to fuzzy image */

  dosharpenpixel<<<nblock, nthread>>>(nx, ny, d, d_conv, d_fuzzyp);
  hipDeviceSynchronize();
  
  /* End of parallel region and convolution computation */
  
  tstop = wtime();
  time = tstop - tstart;
  
  hipMemcpy(convolution, d_conv, nx*ny*sizeof(double),
             hipMemcpyDeviceToHost);

  if (verbose)
    {
      printf("... finished\n");
      printf("\n");
      fflush(stdout);
    }
  
  /* Add rescaled convolution to fuzzy image to obtain sharp image */
  for (i=0; i < nx; i++)
    {
      for (j=0; j < ny; j++)
        {
          sharp[i][j] = fuzzyPadded[i+d][j+d] - scale/norm * convolution[i][j];
        }
    }

  if (verbose)
    {
      printf("Writing output file: %s\n", outfile);
      printf("\n");
    }
  
  /* Only save the core of the sharpened image to remove edge effects */
  for (i=d ; i < nx-d; i++)
    {
      for (j=d; j < ny-d; j++)
        {
          sharpCropped[i-d][j-d] = sharp[i][j];
        }
    }
  
  pgmwrite(outfile, sharpCropped, nx-2*d, ny-2*d);

  if (verbose)
    {
      printf("... done\n");
      printf("\n");
      printf("Calculation time was %f seconds\n", time);
      fflush(stdout);
    }

  // Free memory
  hipFree(d_conv);
  hipFree(d_fuzzyp);
}
